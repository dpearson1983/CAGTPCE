#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <cmath>
#include <hip/hip_vector_types.h>
#include "include/correlation.h"
#include "include/file_io.h"
#include "include/harppi.h"

int main(int argc, char *argv[]) {
    parameters p(argv[1]);
    p.print();
    
    std::cout << "Initial setup..." << std::endl;
    float3 L = {(float)p.getd("Lx"), (float)p.getd("Ly"), (float)p.getd("Lz")};
    float R = (float)p.getd("R");
    int N_shells = p.geti("N_shells");
    
    std::vector<int3> shifts = get_shifts();
    
    hipMemcpyToSymbol(HIP_SYMBOL(d_shifts), shifts.data(), shifts.size()*sizeof(int3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_R), &R, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_L), &L, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(d_Nshells), &N_shells, sizeof(int));
    
    int3 N = {int(L.x/R), int(L.y/R), int(L.z/R)};
    float3 r_min = {(float)p.getd("r_minx"), (float)p.getd("r_miny"), (float)p.getd("r_minz")};
    
    std::cout << "Reading in galaxies and randoms..." << std::endl;
    std::vector<std::vector<float3>> gals;
    std::vector<std::vector<float3>> rans;
    std::vector<float3> gs;
    std::vector<float3> rs;
    
    fileType galFileType = set_fileType(p.gets("galFileType"));
    fileType ranFileType = set_fileType(p.gets("ranFileType"));
    int num_gals = read_file(p.gets("gal_file"), galFileType, gals, L, R, r_min);
    int num_rans = read_file(p.gets("ran_file"), ranFileType, rans, L, R, r_min);
    std::cout << "num_gals = " << num_gals << std::endl;
    std::cout << "num_rans = " << num_rans << std::endl;
    std::cout << "gals.size() = " << gals.size() << std::endl;
    std::cout << "rans.size() = " << rans.size() << std::endl;
    hipMemcpyToSymbol(HIP_SYMBOL(d_Nparts), &num_gals, sizeof(int));
    
    std::cout << "Setting up storage..." << std::endl;
    int Nshells3 = N_shells*N_shells*N_shells;
    std::vector<int> DD(N_shells), DR(N_shells), DDD(Nshells3), DDR(Nshells3), DRR(Nshells3), RRR(Nshells3);
    std::vector<int> galSizes, ranSizes;
    int *d_DD, *d_DR, *d_DDD, *d_DDR, *d_DRR, *d_RRR;
    float3 **d_gals, **d_rans;
    int *d_galSizes, *d_ranSizes;
    float3 *d_gs, *d_rs;
    
    float3 **h_gals = (float3 **)malloc(gals.size()*sizeof(float3 *)); 
    float3 **h_rans = (float3 **)malloc(rans.size()*sizeof(float3 *));
    for (int i = 0; i < gals.size(); ++i) {
        galSizes.push_back(gals[i].size());
        ranSizes.push_back(rans[i].size());
        hipMalloc((void **)&h_gals[i], gals[i].size()*sizeof(float3));
        hipMemcpy(h_gals[i], gals[i].data(), gals[i].size()*sizeof(float3), hipMemcpyHostToDevice);
        hipMalloc((void **)&h_rans[i], rans[i].size()*sizeof(float3));
        hipMemcpy(h_rans[i], rans[i].data(), rans[i].size()*sizeof(float3), hipMemcpyHostToDevice);
        for (int j = 0; j < gals[i].size(); ++j)
            gs.push_back(gals[i][j]);
        for (int j = 0; j < rans[i].size(); ++j)
            rs.push_back(rans[i][j]);
    }
    hipMalloc(&d_gals, gals.size()*sizeof(float3 *));
    hipMemcpy(d_gals, h_gals, gals.size()*sizeof(float3 *), hipMemcpyHostToDevice);
    hipMalloc(&d_rans, rans.size()*sizeof(float3 *));
    hipMemcpy(d_rans, h_rans, rans.size()*sizeof(float3 *), hipMemcpyHostToDevice);
    
    hipMalloc((void **)&d_DD, DD.size()*sizeof(int));
    hipMalloc((void **)&d_DR, DR.size()*sizeof(int));
    hipMalloc((void **)&d_DDD, DDD.size()*sizeof(int));
    hipMalloc((void **)&d_DDR, DDR.size()*sizeof(int));
    hipMalloc((void **)&d_DRR, DRR.size()*sizeof(int));
    hipMalloc((void **)&d_RRR, RRR.size()*sizeof(int));
    hipMalloc((void **)&d_galSizes, galSizes.size()*sizeof(int));
    hipMalloc((void **)&d_ranSizes, ranSizes.size()*sizeof(int));
    hipMalloc((void **)&d_gs, gs.size()*sizeof(float3));
    hipMalloc((void **)&d_rs, rs.size()*sizeof(float3));
    
    hipMemcpy(d_DD, DD.data(), DD.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_DR, DR.data(), DR.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_DDD, DDD.data(), DDD.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_DDR, DDR.data(), DDR.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_DRR, DRR.data(), DRR.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_RRR, RRR.data(), RRR.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_galSizes, galSizes.data(), galSizes.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ranSizes, ranSizes.data(), ranSizes.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_gs, gs.data(), gs.size()*sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_rs, rs.data(), rs.size()*sizeof(float3), hipMemcpyHostToDevice);
    
    int num_blocks = num_gals/N_threads + 1;
    std::cout << "Two point function..." << std::endl;
    countPairs<<<num_blocks, N_threads>>>(d_gs, d_gals, d_galSizes, d_DD, N);
    countPairs<<<num_blocks, N_threads>>>(d_gs, d_rans, d_ranSizes, d_DR, N);
    
    std::cout << "Three point function..." << std::endl;
    std::cout << "   DDD..." << std::endl;
    countTriangles<<<num_blocks, N_threads>>>(d_gs, d_gals, d_gals, d_galSizes, d_galSizes, d_DDD, N);
    std::cout << "   DDR..." << std::endl;
    countTriangles<<<num_blocks, N_threads>>>(d_gs, d_gals, d_rans, d_galSizes, d_ranSizes, d_DDR, N);
    std::cout << "   DRR..." << std::endl;
    countTriangles<<<num_blocks, N_threads>>>(d_gs, d_rans, d_rans, d_ranSizes, d_ranSizes, d_DRR, N);
    std::cout << "   RRR..." << std::endl;
    num_blocks = num_rans/N_threads + 1;
    hipMemcpyToSymbol(HIP_SYMBOL(d_Nparts), &num_rans, sizeof(int));
    countTriangles<<<num_blocks, N_threads>>>(d_rs, d_rans, d_rans, d_ranSizes, d_ranSizes, d_RRR, N);
    
    hipMemcpy(DD.data(), d_DD, DD.size()*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(DR.data(), d_DR, DR.size()*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(DDD.data(), d_DDD, DDD.size()*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(DDR.data(), d_DDR, DDR.size()*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(DRR.data(), d_DRR, DRR.size()*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(RRR.data(), d_RRR, RRR.size()*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    write_2point_file(p.gets("twoPointFile"), DD, DR, num_gals, num_rans, R, N_shells);
    write_triangle_file(p.gets("threePointFile"), DDD, DDR, DRR, RRR, R, N_shells);
    
    num_blocks = num_gals/N_threads + 1;
    double n_bar = num_gals/(L.x*L.y*L.z);
    double Delta_r = R/N_shells;
    for (int i = 0; i < DDR.size(); ++i)
        DDR[i] = 0;
    hipMemcpy(d_DDR, DDR.data(), DDR.size()*sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    getDDR<<<num_blocks, N_threads>>>(d_gs, d_gals, d_galSizes, d_DDR, N, n_bar);
    hipMemcpy(DDR.data(), d_DDR, DDR.size()*sizeof(int), hipMemcpyDeviceToHost);
    
    RRR = getRRR(Delta_r, n_bar, num_rans, N_shells);
    
    write_triangle_file("predicted.dat", DDD, DDR, RRR, RRR, R, N_shells);
    
    hipFree(d_DD);
    hipFree(d_DR);
    hipFree(d_DDD);
    hipFree(d_DDR);
    hipFree(d_DRR);
    hipFree(d_RRR);
    hipFree(d_galSizes);
    hipFree(d_ranSizes);
    hipFree(d_gals);
    hipFree(d_rans);
    hipFree(d_gs);
    hipFree(d_rs);
    delete[] h_gals;
    delete[] h_rans;
    
    return 0;
}
    
    
    
